#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "mpi.h"

constexpr int _PageSize = 4096;

#define CUDACHECK(cmd)                                              \
  do {                                                              \
    hipError_t e = cmd;                                            \
    if (e != hipSuccess) {                                         \
      printf("Failed: Cuda error %s:%d '%s'\n", __FILE__, __LINE__, \
             hipGetErrorString(e));                                \
      exit(EXIT_FAILURE);                                           \
    }                                                               \
  } while (0)

#define MPICHECK(cmd)                                                  \
  do {                                                                 \
    int e = cmd;                                                       \
    if (e != MPI_SUCCESS) {                                            \
      printf("Failed: MPI error %s:%d '%d'\n", __FILE__, __LINE__, e); \
      exit(EXIT_FAILURE);                                              \
    }                                                                  \
  } while (0)

__global__ void initPeerMemory(int* dev_ptr) {
  int tid = threadIdx.x;
  *(dev_ptr + tid) = tid;
}

__global__ void loadPeerMemory(int* dev_ptr, int* peer_dev_ptr) {
  int tid = threadIdx.x;
  int* peer_ptr = peer_dev_ptr + tid;
  
  // First normal load, cache data
  int val = *peer_ptr;
  val *= 2;

  // Second load
#ifdef ENABLE_VOLATILE
  volatile int* volatile_peer_ptr = (volatile int*)peer_ptr;
  int val_new = *volatile_peer_ptr;
#elif ENABLE_LDCV
  int val_new = __ldcv(peer_ptr);
#else
  int val_new = *peer_ptr;
#endif

  val += val_new;
  *(dev_ptr + tid) = val;
}

int run(int rank) {
  int* dev_ptr;
  hipIpcMemHandle_t ipc_memory_handle;
  hipIpcMemHandle_t ipc_memory_handle_array[2];
  CUDACHECK(hipMalloc(&dev_ptr, _PageSize));
  CUDACHECK(hipMemset(dev_ptr, 0, _PageSize));
  // Init memory handle
  CUDACHECK(hipIpcGetMemHandle(&ipc_memory_handle, dev_ptr));
  MPICHECK(MPI_Allgather(&ipc_memory_handle, sizeof(hipIpcMemHandle_t),
    MPI_BYTE, ipc_memory_handle_array, sizeof(hipIpcMemHandle_t), MPI_BYTE, MPI_COMM_WORLD));

  // Get peer device pointer
  int* peer_dev_ptr;
  if (rank == 0) {
    CUDACHECK(hipIpcOpenMemHandle((void**)&peer_dev_ptr, ipc_memory_handle_array[1], hipIpcMemLazyEnablePeerAccess));
  } else {
    CUDACHECK(hipIpcOpenMemHandle((void**)&peer_dev_ptr, ipc_memory_handle_array[0], hipIpcMemLazyEnablePeerAccess));
  }

  if (rank == 1) {
    dim3 grid(1, 1, 1);
    dim3 block(32, 1, 1);
    initPeerMemory<<<grid, block>>>(dev_ptr);
    CUDACHECK(hipDeviceSynchronize());
  }

  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));

  if (rank == 0) {
    dim3 grid(1, 1, 1);
    dim3 block(32, 1, 1);
    loadPeerMemory<<<grid, block>>>(dev_ptr, peer_dev_ptr);
    CUDACHECK(hipDeviceSynchronize());
    int* host_ptr = (int*)malloc(_PageSize);
    CUDACHECK(hipMemcpy(host_ptr, dev_ptr, _PageSize, hipMemcpyDeviceToHost));
    printf("3x Peer Values: ");
    for (int i = 0; i < 32; i++) {
      printf("%d  ", *(host_ptr + i));
    }
    printf("\n");
    free(host_ptr);
  }
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));

  CUDACHECK(hipFree(dev_ptr));
  return 0;
}

int main(int argc, char** argv) {
  int nRanks, myRank;
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));
  if (nRanks != 2) {
    printf("Error: Wrong World Size %d\n", nRanks);
    MPICHECK(MPI_Finalize());
    return 0;
  }
  CUDACHECK(hipSetDevice(myRank));
  run(myRank);
  MPICHECK(MPI_Finalize());
  return 0;
}
